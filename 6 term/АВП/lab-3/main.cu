
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <stdlib.h>
#include <time.h>

#define N 15
#define M 1


void TransformArrayHost(const short *input_array, short **result_array) {
    for (int i = 0; i < N; i++)
        result_array[i][0] = input_array[i];
}


__global__ void TransformArrayDevice(const short *input_array, short **result_array_dev_dev) {
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        result_array_dev_dev[i][0] = input_array[i];
}


bool Compare(short **gpu_array, short **cpu_array) {
    auto temp = new short[M];
    for (int i = 0; i < N; i++) {
        hipMemcpy(temp, gpu_array[i], M * sizeof(short), hipMemcpyDeviceToHost);
        if (temp[0] != cpu_array[i][0])
            return false;
    }
    return true;
}


int main() {
    srand(time(NULL));
    auto *input_array = new short[N];

    auto **result_array = new short *[N];
    for (int i = 0; i < N; i++)
        result_array[i] = new short[M];

    // Allocate memory for 2D array
    short *result_array_dev_flat;
    hipMalloc((void **) &result_array_dev_flat, N * M * sizeof(short));
    auto **result_array_dev = new short *[N];
    for (int i = 0; i < N; i++)
        result_array_dev[i] = &result_array_dev_flat[i * M];
    short **result_array_dev_dev;
    hipMalloc((void **) &result_array_dev_dev, N * sizeof(short *));
    hipMemcpy(result_array_dev_dev, result_array_dev, N * sizeof(short *), hipMemcpyHostToDevice);

    // Fill input array with values
    for (int i = 0; i < N; i++)
        input_array[i] = rand() % 100;

    // Create input_array for device
    short *input_array_dev;
    hipMalloc((void **) &input_array_dev, N * sizeof(short));
    hipMemcpy(input_array_dev, input_array, N * sizeof(short), hipMemcpyHostToDevice);

    std::cout << "--------------Source Array--------------" << std::endl;
    for (int i = 0; i < N; i++)
        printf("%d ", input_array[i]);
    std::cout << std::endl;

    // CPU processing
    auto t1 = std::chrono::high_resolution_clock::now();
    TransformArrayHost(input_array, result_array);
    auto t2 = std::chrono::high_resolution_clock::now();
    std::cout << "--------------CPU Array--------------" << std::endl;
    std::cout << ((std::chrono::duration<double>) (t2 - t1)).count() << std::endl;
    for (int i = 0; i < N; i++)
        printf("%d\n", result_array[i][0]);

    // GPU processing
    t1 = std::chrono::high_resolution_clock::now();
    dim3 threadsPerBlock(512);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x);
    TransformArrayDevice<<<numBlocks, threadsPerBlock>>>(input_array_dev, result_array_dev_dev);
    t2 = std::chrono::high_resolution_clock::now();

    hipMemcpy(result_array_dev, result_array_dev_dev, sizeof(short *) * N, hipMemcpyDeviceToHost);
    auto temp = new short[M];
    std::cout << "--------------GPU Array--------------" << std::endl;
    std::cout << ((std::chrono::duration<double>) (t2 - t1)).count() << std::endl;
    for (int i = 0; i < N; i++) {
        hipMemcpy(temp, result_array_dev[i], M * sizeof(short), hipMemcpyDeviceToHost);
        std::cout << temp[0] << std::endl;
    }
    if (Compare(result_array_dev, result_array))
        std::cout << "CPU and GPU arrays are equal" << std::endl;
    else
        std::cout << "CPU and GPU arrays aren't equal" << std::endl;

    hipFree(input_array_dev);
    hipFree(result_array_dev_dev);
    hipFree(result_array_dev_flat);
    delete[] temp;
    delete[] input_array;
    for (int i = 0; i < N; i++)
        delete[] result_array[i];
    delete[] result_array;

    return 0;
}
