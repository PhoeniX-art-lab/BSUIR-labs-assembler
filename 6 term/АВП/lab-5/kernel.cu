#include "hip/hip_runtime.h"
﻿#ifndef CUDACC_RTC
#define CUDACC_RTC
#endif
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "hip/device_functions.h"
#include <opencv2/opencv.hpp>
#include <opencv2/photo/cuda.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/core/cuda.hpp>
#include<assert.h>
#include <stdio.h>
#include <cmath>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <chrono>
#include <hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<vector>
#include<numeric>

using namespace std;
using namespace std::chrono;
#define AMOUNT 100000000
#define GRID_SIZE 128

using namespace cv;
using namespace std;

__global__ void thresholdImageKernel(unsigned char* colorData, unsigned char* thresholdData,
	int colorPitch, int thresholdPitch, int rows, int cols) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < rows && col < cols) {
		// Вычисление указателя на текущий пиксель в буфере цветовых данных
		unsigned char* rowPtr = colorData + row * colorPitch;
		unsigned char color = rowPtr[col * 3];	// Получение значения цвета для текущего пикселя

		// Приведение значения цвета к 0 или 255 и запись результата в выходной буфер
		thresholdData[row * thresholdPitch + col] = (color == 255) ? 255 : 0;
	}
}

__global__ void HoughTransformKernel(unsigned char* src, size_t rows, size_t cols, size_t pitch, int* accumulator,
	float diagonal)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < rows && col < cols) {
		if (*(src + pitch * row + col) > 0)	// Проверка, является ли пиксель белым
		{
			// Перебор всех возможных углов для поиска линии на входном изображении
			for (int t = 0; t < 180; t++)
			{
				// Вычисление расстояния между текущим пикселем и линией под текущим углом
				float r = (float)(col * cosf(t * CV_PI / 180) + row * sinf(t * CV_PI / 180));
				// Вычисляем индекс в массиве аккумуляторов для этой строки
				int irho = int(r + diagonal / 2);

				// Добавляем угол для данной строки в массив аккумуляторов
				atomicAdd(accumulator + 180 * irho + t, 1);
			}
		}
	}
}

__global__ void rotateImageKernel(uchar* src, uchar* dst, int srcPitch, int dstPitch, int cols, int rows, double angle, int channels,
	int centerX, int centerY, double sinAngle, double cosAngle)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (y < rows && x < cols)
	{	
		// Вычисляем повернутые координаты xRotated и yRotated для каждого потока, используя центр вращения и угол
		double xRotated = (x - centerX) * cosAngle - (y - centerY) * sinAngle + centerX;
		double yRotated = (x - centerX) * sinAngle + (y - centerY) * cosAngle + centerY;

		// Вычисляем четыре окружающих пикселя x1, x2, y1 и y2 вокруг повернутых координат
		int x1 = floor(xRotated);
		int x2 = ceil(xRotated);
		int y1 = floor(yRotated);
		int y2 = ceil(yRotated);

		// Вычисляем расстояние между повернутыми координатами и окружающими пикселями
		double dx1 = xRotated - x1;
		double dx2 = x2 - xRotated;
		double dy1 = yRotated - y1;
		double dy2 = y2 - yRotated;

		// Прижимаем окружающие пиксели к границе изображения
		x1 = max(0, min(x1, cols - 1));
		x2 = max(0, min(x2, cols - 1));
		y1 = max(0, min(y1, rows - 1));
		y2 = max(0, min(y2, rows - 1));

		for (int c = 0; c < channels; c++)
		{
			// Вычисляем средневзвешенное значение четырех окружающих пикселей по расстояниям dx, dy
			double value = src[y1 * srcPitch + x1 * channels + c] * dx2 * dy2
				+ src[y1 * srcPitch + x2 * channels + c] * dx1 * dy2
				+ src[y2 * srcPitch + x1 * channels + c] * dx2 * dy1
				+ src[y2 * srcPitch + x2 * channels + c] * dx1 * dy1;
			dst[y * dstPitch + x * channels + c] = static_cast<uchar>(value); // Сохраняем результат в изображении
		}
	}
}

int main(int argc, char** argv)
{

	//Mat h_src(500, 500, CV_8UC3, Scalar(255, 255, 255));
	Mat h_src = cv::imread("/YourPath", cv::IMREAD_COLOR);
	float angle = (float)(rand() % 360 - 180) * CV_PI / 180.0;
	float x_start = h_src.cols / 2; // определение точки центра изображения по горизонтали
	float y_start = h_src.rows / 2; // определение точки центра изображения по вертикали
	line(h_src,
		Point(x_start, y_start),
		Point(x_start + 500 * cos(angle), y_start + 500 * sin(angle)),
		Scalar(255, 0, 0),
		6);
	imshow("source", h_src);
	waitKey(0);

	unsigned char* colorData = h_src.ptr<unsigned char>(); // получаем указатель на пиксельное изображение
	size_t rows = h_src.rows; // количество строк в изображении
	size_t cols = h_src.cols; // количество столбцов в изображении

	// Выделяем память на устройстве для указателей на цветное и пороговое изображение
	uchar* d_colorData;
	uchar* d_thresholdData;

	size_t colorPitch;
	hipMallocPitch((void**)&d_colorData, &colorPitch, cols * 3 * sizeof(unsigned char), rows);

	size_t threshholdPitch;
	hipMallocPitch((void**)&d_thresholdData, &threshholdPitch, cols * sizeof(unsigned char), rows);

	hipMemcpy2D(d_colorData, colorPitch, colorData, cols * 3 *
		sizeof(unsigned char),
		cols * 3 * sizeof(unsigned char), rows, hipMemcpyHostToDevice);
	dim3 block(16, 16);
	dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
	thresholdImageKernel <<<grid, block>>>(d_colorData, d_thresholdData, colorPitch,
		threshholdPitch, rows, cols);
	


	uchar* test = new unsigned char[rows * cols * 3];
	hipMemcpy2D(test, cols * 3 * sizeof(uchar), d_thresholdData, threshholdPitch,
		cols * sizeof(uchar), rows, hipMemcpyDeviceToHost);

	Mat cv_test(rows, cols, CV_8UC3, test);
	
	imshow("test", cv_test);
	waitKey(0);
 	

	// Вычисляем длину диагонали изображения
	float diagonal = sqrt((float)cols * cols + (float)rows * rows);

	// Выделяем память на устройстве для указателя на массив аккумулятора
	int* d_accumulator;
	hipMalloc((void**)&d_accumulator, 180 * (int)diagonal * sizeof(int));

	HoughTransformKernel <<<grid, block >>>(d_thresholdData, rows, cols, threshholdPitch, d_accumulator,
		diagonal);

	// Копируем массив аккумулятора с девайса на хост
	int* accumulator = new int[180 * (int)diagonal];
	hipMemcpy(accumulator, d_accumulator, 180 * (int)diagonal * sizeof(int), hipMemcpyDeviceToHost);

	// Создаем вектор линий
	std::vector<cv::Vec2f> lines;
	// Проходимся по каждой строке и столбцу в аккумуляторе, чтобы найти линии
	for (int r = 0; r < (int)diagonal; r++)
	{
		for (int t = 0; t < 180; t++)
		{
			if (accumulator[180 * r + t] >= 500)	// значение в аккумуляторе больше порогового
			{	
				// Сохраняем найденную линию
				cv::Vec2f line(r - (int)diagonal / 2, t);
				lines.push_back(line);

				if (lines.size() >= 20)	// если нашли достаточно линий
				{
					break;
				}
			}
		}
	}

	// Выводим координаты линий
	for (auto item : lines)
	{
		std::cout << item << std::endl;
	}
	cout << endl << endl << lines.size();

	uchar* d_result;
	size_t resultPitch;
	double sinAngle = sin(lines[0][1] * CV_PI / 180);
	double cosAngle = cos(lines[0][1] * CV_PI / 180);

	hipMallocPitch((void**)&d_result, &resultPitch, cols * 3 * sizeof(unsigned char), rows);

	rotateImageKernel << <grid, block >> >(d_colorData, d_result, colorPitch, resultPitch, cols, rows, lines[0][1] * CV_PI / 180, 3,
		cols / 2, rows / 2, sinAngle, cosAngle);

	uchar* result = new unsigned char[rows * cols * 3];
	hipMemcpy2D(result, cols * 3 * sizeof(uchar), d_result, resultPitch,
		cols * 3 * sizeof(unsigned char), rows, hipMemcpyDeviceToHost);

	Mat cv_dst(rows, cols, CV_8UC3, result);

	imshow("result", cv_dst);
	waitKey(0);

	destroyAllWindows();
	return 0;
}
